#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <math.h>

__global__ void cpd_psumpt1_cuda(int m, int n, float thresh, float expFactor, float denomAdd, float* ctx);
__global__ void cpd_p1px_cuda(int m, int n, float thresh, float expFactor, float* ctx);
__global__ void cpd_sigmaest_cuda(int m, int n, float* ctx);

#define BLOCK_SIZE (512)

bool cpd_init_cuda(int m, int n, const void* x, void** ppDevCtx)
{
    // Layout:
    size_t devMemorySize = sizeof(float) * (3 * n + 3 * m + n + m + 3 * m);

    if (hipMalloc(ppDevCtx, devMemorySize) != hipSuccess)
        return false;

    float* dx = (float*)*ppDevCtx;
    hipMemcpy(dx, x, sizeof(float) * 3 * n, hipMemcpyHostToDevice);

    return true;
}

void cpd_deinit_cuda(void* pDevCtx)
{
    hipFree(pDevCtx);
}

float cpd_estimate_sigma_cuda(void* pDevCtx, const float* x, const float* t, int m, int n)
{
    float* dx = (float*)pDevCtx;
    float* dt = dx + 3 * n;
    float* dtemp = dt + 3 * m;

    hipMemcpy(dt, t, sizeof(float) * 3 * m, hipMemcpyHostToDevice);
    hipMemset(dtemp, 0, sizeof(float) * n);

    const int threadsPerBlock = BLOCK_SIZE;

    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    cpd_sigmaest_cuda<<<blocksPerGrid, threadsPerBlock>>> (m, n, dx);

    float* sumpart = new float[n];
    hipMemcpy(sumpart, dtemp, sizeof(float) * n, hipMemcpyDeviceToHost);

    float sum = 0;
    for (int i = 0; i < n; i++)
        sum += sumpart[i];

    delete[] sumpart;

    return sum / (3 * m * n);
}

void cpd_estep_cuda(void* pDevCtx, const float* x, const float* t, int m, int n, float w, float sigma2, float denom, float* pt1p1px)
{
    const float factor = -1.0f / (2.0f * sigma2);
    const float thresh = std::max(0.0001f, 2.0f * sqrtf(sigma2));

    float* dx = (float*)pDevCtx;
    float* dt = dx + 3 * n;
    float* dpt1 = dt + 3 * m; // dpt1, dp1, dpx must be in the correct sequence, we'll be copying them in one operation
    float* dp1 = dpt1 + n;
    float* dpx = dp1 + m;  
    //float* dpsum = dpx + 3 * m;
    hipMemcpy(dt, t, sizeof(float) * 3 * m, hipMemcpyHostToDevice);
    hipMemset(dpx, 0, sizeof(float) * (3 * m + m + n + n));

    const int threadsPerBlock = BLOCK_SIZE;

    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    cpd_psumpt1_cuda<<<blocksPerGrid, threadsPerBlock>>>(m, n, thresh, factor, denom, dx);

    blocksPerGrid = (m + threadsPerBlock - 1) / threadsPerBlock;
    cpd_p1px_cuda<<<blocksPerGrid, threadsPerBlock>>>(m, n, thresh, factor, dx);
   
    // TODO: check errors
    hipMemcpy(pt1p1px, dpt1, sizeof(float) * (n + m + 3 * m), hipMemcpyDeviceToHost);
}

__global__ void cpd_psumpt1_cuda(int m, int n, float thresh, float expFactor, float denomAdd, float* ctx)
{
    __shared__ float t012[4 * BLOCK_SIZE];

    int thread = threadIdx.x;
    int i = thread + blockIdx.x * BLOCK_SIZE;

    float* x = ctx;
    float* t = x + 3 * n;
    float* pt1 = t + 3 * m;
    float* p1 = pt1 + n;
    float* px = p1 + m;
    float* psum = px + 3 * m;

    float sum = 0;
    float x0 = 0, x1 = 0, x2 = 0;
    if (i < n) {
        x0 = x[0 * n + i];
        x1 = x[1 * n + i];
        x2 = x[2 * n + i];
    }

    for (int jb = 0; jb < m; jb += BLOCK_SIZE) {
        int mb = __min(m, jb + BLOCK_SIZE);

        int jthread = jb + thread;
        t012[4 * thread + 0] = t[0 * m + jthread];
        t012[4 * thread + 1] = t[1 * m + jthread];
        t012[4 * thread + 2] = t[2 * m + jthread];
        // 4th element is a dummy

        __syncthreads();

        if (i < n) {
            for (int j = jb; j < mb; j++) {
                int jj = j - jb;

                float d0 = x0 - t012[4 * jj + 0];
                float d1 = x1 - t012[4 * jj + 1];
                float d2 = x2 - t012[4 * jj + 2];
                float dist = __fmaf_rz(d0, d0, __fmaf_rz(d1, d1, __fmul_rz(d2, d2)));

                if (dist < thresh)
                    sum += __expf(expFactor * dist);
            }
        }

        __syncthreads();
    }

    if (i < n) {
        psum[i] = 1.0f / (sum + denomAdd);
        pt1[i] = sum / (sum + denomAdd);
    }
}

__global__ void cpd_p1px_cuda(int m, int n, float thresh, float expFactor, float* ctx)
{
    __shared__ float x012sum[4 * BLOCK_SIZE];

    int thread = threadIdx.x;
    int j = thread + blockIdx.x * BLOCK_SIZE;

    float* x = ctx;
    float* t = x + 3 * n;
    float* pt1 = t + 3 * m;
    float* p1 = pt1 + n;
    float* px = p1 + m;
    float* psum = px + 3 * m;

    float sumpx0 = 0, sumpx1 = 0, sumpx2 = 0;
    float sump1 = 0.0f;

    float t0 = 0, t1 = 0, t2 = 0;
    if (j < m) {
        t0 = t[0 * m + j];
        t1 = t[1 * m + j];
        t2 = t[2 * m + j];
    }

    for (int ib = 0; ib < n; ib += BLOCK_SIZE) {
        int nb = __min(n, ib + BLOCK_SIZE);

        int ithread = ib + thread;
        x012sum[4 * thread + 0] = x[0 * n + ithread];
        x012sum[4 * thread + 1] = x[1 * n + ithread];
        x012sum[4 * thread + 2] = x[2 * n + ithread];
        x012sum[4 * thread + 3] = psum[ithread];

        __syncthreads();

        if (j < m) {
            for (int i = ib; i < nb; i++) {
                int ii = i - ib;

                float d0 = x012sum[4 * ii + 0] - t0;
                float d1 = x012sum[4 * ii + 1] - t1;
                float d2 = x012sum[4 * ii + 2] - t2;
                float dist = __fmaf_rz(d0, d0, __fmaf_rz(d1, d1, __fmul_rz(d2, d2)));

                if (dist < thresh) {
                    float pmn = __expf(expFactor * dist) * x012sum[4 * ii + 3];
                    sumpx0 += pmn * x012sum[4 * ii + 0];
                    sumpx1 += pmn * x012sum[4 * ii + 1];
                    sumpx2 += pmn * x012sum[4 * ii + 2];
                    sump1 += pmn;
                }
            }
        }

        __syncthreads();
    }

    if (j < m) {
        p1[j] = sump1;
        px[0 * m + j] = sumpx0;
        px[1 * m + j] = sumpx1;
        px[2 * m + j] = sumpx2;
    }
}

__global__ void cpd_sigmaest_cuda(int m, int n, float* ctx)
{
    int i = threadIdx.x + blockIdx.x * BLOCK_SIZE;

    if (i < n) {
        float* x = ctx;
        float* t = x + 3 * n;
        float* temp = t + 3 * m;

        float accum = 0.0f;
        for (int j = 0; j < m; j++) {
            float dd0 = x[0 * n + i] - t[0 * m + j];
            float dd1 = x[1 * n + i] - t[1 * m + j];
            float dd2 = x[2 * n + i] - t[2 * m + j];
            float dist = dd0 * dd0 + dd1 * dd1 + dd2 * dd2;
            accum += dist;
        }

        temp[i] = accum;
    }
}
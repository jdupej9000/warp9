#include <hip/hip_runtime.h>


#include <math.h>

__global__ void cpd_psumpt1_cuda(int m, int n, float thresh, float expFactor, float denomAdd, float* ctx);
__global__ void cpd_p1px_cuda(int m, int n, float thresh, float expFactor, float* ctx);
__global__ void cpd_sigmaest_cuda(int m, int n, float* ctx);

bool cpd_init_cuda(int m, int n, const void* x, void** ppDevCtx)
{
    // Layout:
    size_t devMemorySize = sizeof(float) * (3 * n + 3 * m + n + m + 3 * m);

    if (hipMalloc(ppDevCtx, devMemorySize) != hipSuccess)
        return false;

    float* dx = (float*)*ppDevCtx;
    hipMemcpy(dx, x, sizeof(float) * 3 * n, hipMemcpyHostToDevice);

    return true;
}

void cpd_deinit_cuda(void* pDevCtx)
{
    hipFree(pDevCtx);
}

float cpd_estimate_sigma_cuda(void* pDevCtx, const float* x, const float* t, int m, int n)
{
    float* dx = (float*)pDevCtx;
    float* dt = dx + 3 * n;
    float* dtemp = dt + 3 * m;

    hipMemcpy(dt, t, sizeof(float) * 3 * m, hipMemcpyHostToDevice);
    hipMemset(dtemp, 0, sizeof(float) * n);

    const int threadsPerBlock = 512;

    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    cpd_sigmaest_cuda<<<blocksPerGrid, threadsPerBlock>>> (m, n, dx);

    float* sumpart = new float[n];
    hipMemcpy(sumpart, dtemp, sizeof(float) * n, hipMemcpyDeviceToHost);

    float sum = 0;
    for (int i = 0; i < n; i++)
        sum += sumpart[i];

    delete[] sumpart;

    return sum / (3 * m * n);
}

void cpd_estep_cuda(void* pDevCtx, const float* x, const float* t, int m, int n, float w, float sigma2, float denom, float* pt1p1px)
{
    const float factor = -1.0f / (2.0f * sigma2);
    const float thresh = std::max(0.0001f, 2.0f * sqrtf(sigma2));

    float* dx = (float*)pDevCtx;
    float* dt = dx + 3 * n;
    float* dpt1 = dt + 3 * m; // dpt1, dp1, dpx must be in the correct sequence, we'll be copying them in one operation
    float* dp1 = dpt1 + n;
    float* dpx = dp1 + m;  
    //float* dpsum = dpx + 3 * m;
    hipMemcpy(dt, t, sizeof(float) * 3 * m, hipMemcpyHostToDevice);
    hipMemset(dpx, 0, sizeof(float) * (3 * m + m + n + n));

    const int threadsPerBlock = 512;

    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    cpd_psumpt1_cuda<<<blocksPerGrid, threadsPerBlock>>>(m, n, thresh, factor, denom, dx);

    blocksPerGrid = (m + threadsPerBlock - 1) / threadsPerBlock;
    cpd_p1px_cuda<<<blocksPerGrid, threadsPerBlock>>>(m, n, thresh, factor, dx);
   
    // TODO: check errors
    hipMemcpy(pt1p1px, dpt1, sizeof(float) * (n + m + 3 * m), hipMemcpyDeviceToHost);
}

__global__ void cpd_psumpt1_cuda(int m, int n, float thresh, float expFactor, float denomAdd, float* ctx)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < n) {
        float* x = ctx;
        float* t = x + 3 * n;
        float* pt1 = t + 3 * m;
        float* p1 = pt1 + n;
        float* px = p1 + m;
        float* psum = px + 3 * m;

        float sum = 0;
        const float x0 = x[0 * n + i];
        const float x1 = x[1 * n + i];
        const float x2 = x[2 * n + i];

        for (int j = 0; j < m; j++) {
            float d0 = x0 - t[0 * m + j];
            float d1 = x1 - t[1 * m + j];
            float d2 = x2 - t[2 * m + j];
            float dist = __fmaf_rz(d0, d0, __fmaf_rz(d1, d1, __fmul_rz(d2, d2)));

            if (dist < thresh)
                sum += __expf(expFactor * dist);
        }

        psum[i] = 1.0f / (sum + denomAdd);
        pt1[i] = sum / (sum + denomAdd);
    }
}

__global__ void cpd_p1px_cuda(int m, int n, float thresh, float expFactor, float* ctx)
{
    int j = threadIdx.x + blockIdx.x * blockDim.x;

    if (j < m) {
        float* x = ctx;
        float* t = x + 3 * n;
        float* pt1 = t + 3 * m;
        float* p1 = pt1 + n;
        float* px = p1 + m;
        float* psum = px + 3 * m;

        float sumpx0 = 0, sumpx1 = 0, sumpx2 = 0;
        float sump1 = 0.0f;
        const float t0 = t[0 * m + j];
        const float t1 = t[1 * m + j];
        const float t2 = t[2 * m + j];

        for (int i = 0; i < n; i++) {
            const float x0 = x[0 * n + i];
            const float x1 = x[1 * n + i];
            const float x2 = x[2 * n + i];

            float d0 = x0 - t0;
            float d1 = x1 - t1;
            float d2 = x2 - t2;
            float dist = __fmaf_rz(d0, d0, __fmaf_rz(d1, d1, __fmul_rz(d2, d2)));

            if (dist < thresh) {
                float pmn = __expf(expFactor * dist) * psum[i];
                sumpx0 += pmn * x0;
                sumpx1 += pmn * x1;
                sumpx2 += pmn * x2;
                sump1 += pmn;
            }
        }

        p1[j] = sump1;
        px[0 * m + j] = sumpx0;
        px[1 * m + j] = sumpx1;
        px[2 * m + j] = sumpx2;
    }
}

__global__ void cpd_sigmaest_cuda(int m, int n, float* ctx)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < n) {
        float* x = ctx;
        float* t = x + 3 * n;
        float* temp = t + 3 * m;

        float accum = 0.0f;
        for (int j = 0; j < m; j++) {
            float dd0 = x[0 * n + i] - t[0 * m + j];
            float dd1 = x[1 * n + i] - t[1 * m + j];
            float dd2 = x[2 * n + i] - t[2 * m + j];
            float dist = dd0 * dd0 + dd1 * dd1 + dd2 * dd2;
            accum += dist;
        }

        temp[i] = accum;
    }
}
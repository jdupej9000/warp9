#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <math.h>

#define CONST_ARG const __grid_constant__

__global__ void cpd_psumpt1_cuda(CONST_ARG int m, CONST_ARG int n, CONST_ARG float thresh, CONST_ARG float expFactor, CONST_ARG float denomAdd, float* ctx);
__global__ void cpd_p1px_cuda(CONST_ARG int m, CONST_ARG int n, CONST_ARG float thresh, CONST_ARG float expFactor, float* ctx);
__global__ void cpd_sigmaest_cuda(CONST_ARG int m, CONST_ARG int n, float* ctx);

#define BLOCK_SIZE (512)

bool cpd_init_cuda(int m, int n, const void* x, void** ppDevCtx)
{
    // Layout:
    size_t devMemorySize = sizeof(float) * (3 * n + 3 * m + n + m + 3 * m);

    if (hipMalloc(ppDevCtx, devMemorySize) != hipSuccess)
        return false;

    float* dx = (float*)*ppDevCtx;
    hipMemcpy(dx, x, sizeof(float) * 3 * n, hipMemcpyHostToDevice);

    return true;
}

void cpd_deinit_cuda(void* pDevCtx)
{
    hipFree(pDevCtx);
}

float cpd_estimate_sigma_cuda(void* pDevCtx, const float* x, const float* t, int m, int n)
{
    float* dx = (float*)pDevCtx;
    float* dt = dx + 3 * n;
    float* dtemp = dt + 3 * m;

    hipMemcpy(dt, t, sizeof(float) * 3 * m, hipMemcpyHostToDevice);
    hipMemset(dtemp, 0, sizeof(float) * n);

    const int threadsPerBlock = BLOCK_SIZE;

    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    cpd_sigmaest_cuda<<<blocksPerGrid, threadsPerBlock>>> (m, n, dx);

    float* sumpart = new float[n];
    hipMemcpy(sumpart, dtemp, sizeof(float) * n, hipMemcpyDeviceToHost);

    float sum = 0;
    for (int i = 0; i < n; i++)
        sum += sumpart[i];

    delete[] sumpart;

    return sum / (3 * m * n);
}

void cpd_estep_cuda(void* pDevCtx, const float* x, const float* t, int m, int n, float w, float sigma2, float denom, float* pt1p1px)
{
    const float factor = -1.0f / (2.0f * sigma2);
    const float thresh = std::max(0.0001f, 2.0f * sqrtf(sigma2));

    float* dx = (float*)pDevCtx;
    float* dt = dx + 3 * n;
    float* dpt1 = dt + 3 * m; // dpt1, dp1, dpx must be in the correct sequence, we'll be copying them in one operation
    float* dp1 = dpt1 + n;
    float* dpx = dp1 + m;  
    //float* dpsum = dpx + 3 * m;
    hipMemcpy(dt, t, sizeof(float) * 3 * m, hipMemcpyHostToDevice);
    hipMemset(dpx, 0, sizeof(float) * (3 * m + m + n + n));

    const int threadsPerBlock = BLOCK_SIZE;

    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    cpd_psumpt1_cuda<<<blocksPerGrid, threadsPerBlock>>>(m, n, thresh, factor, denom, dx);

    blocksPerGrid = (m + threadsPerBlock - 1) / threadsPerBlock;
    cpd_p1px_cuda<<<blocksPerGrid, threadsPerBlock>>>(m, n, thresh, factor, dx);
   
    // TODO: check errors
    hipMemcpy(pt1p1px, dpt1, sizeof(float) * (n + m + 3 * m), hipMemcpyDeviceToHost);
}

__global__ void cpd_psumpt1_cuda(CONST_ARG int m, CONST_ARG int n, CONST_ARG float thresh, CONST_ARG float expFactor, CONST_ARG float denomAdd, float* ctx)
{
    __shared__ float t012[3 * BLOCK_SIZE];

    int thread = threadIdx.x;
    int i = thread + blockIdx.x * BLOCK_SIZE;

    float* x = ctx;
    float* t = x + 3 * n;
    float* pt1 = t + 3 * m;
    float* p1 = pt1 + n;
    float* px = p1 + m;
    float* psum = px + 3 * m;

    float sum = 0;
    float x0 = 0, x1 = 0, x2 = 0;
    if (i < n) {
        x0 = x[0 * n + i];
        x1 = x[1 * n + i];
        x2 = x[2 * n + i];
    }

    for (int jb = 0; jb < m; jb += BLOCK_SIZE) {
        int mb = __min(m, jb + BLOCK_SIZE) - jb;

        int jthread = jb + thread;
        t012[3 * thread + 0] = t[0 * m + jthread];
        t012[3 * thread + 1] = t[1 * m + jthread];
        t012[3 * thread + 2] = t[2 * m + jthread];

        __syncthreads();

        if (i < n) {
            float sumb = 0;
            for (int j = 0; j < mb; j++) {
                float d0 = x0 - t012[3 * j + 0];
                float d1 = x1 - t012[3 * j + 1];
                float d2 = x2 - t012[3 * j + 2];
                float dist = fmaf(d0, d0, fmaf(d1, d1, d2 * d2));

                if (dist < thresh)
                    sumb += __expf(expFactor * dist);
            }
            sum += sumb;
        }

        __syncthreads();
    }

    if (i < n) {
        if (fabs(sum + denomAdd) > 1e-5f) {
            float psumi = 1.0f / (sum + denomAdd);
            psum[i] = psumi;
            pt1[i] = sum * psumi;
        }
        else {
            psum[i] = 10000;
            pt1[i] = 0;
        }
    }
}

__global__ void cpd_p1px_cuda(CONST_ARG int m, CONST_ARG int n, CONST_ARG float thresh, CONST_ARG float expFactor, float* ctx)
{
    __shared__ float x012sum[4 * BLOCK_SIZE];

    int thread = threadIdx.x;
    int j = thread + blockIdx.x * BLOCK_SIZE;

    float* x = ctx;
    float* t = x + 3 * n;
    float* pt1 = t + 3 * m;
    float* p1 = pt1 + n;
    float* px = p1 + m;
    float* psum = px + 3 * m;

    float sumpx0 = 0, sumpx1 = 0, sumpx2 = 0;
    float sump1 = 0.0f;

    float t0 = 0, t1 = 0, t2 = 0;
    if (j < m) {
        t0 = t[0 * m + j];
        t1 = t[1 * m + j];
        t2 = t[2 * m + j];
    }

    for (int ib = 0; ib < n; ib += BLOCK_SIZE) {
        int nb = __min(n, ib + BLOCK_SIZE) - ib;

        int ithread = ib + thread;
        x012sum[4 * thread + 0] = x[0 * n + ithread];
        x012sum[4 * thread + 1] = x[1 * n + ithread];
        x012sum[4 * thread + 2] = x[2 * n + ithread];
        x012sum[4 * thread + 3] = psum[ithread];

        __syncthreads();

        if (j < m) {
            float sumpx0b = 0, sumpx1b = 0, sumpx2b = 0, sump1b = 0;
            for (int i = 0; i < nb; i++) {
                float d0 = x012sum[4 * i + 0] - t0;
                float d1 = x012sum[4 * i + 1] - t1;
                float d2 = x012sum[4 * i + 2] - t2;
                float dist = fmaf(d0, d0, fmaf(d1, d1, d2 * d2));

                if (dist < thresh) {
                    float pmn = __expf(expFactor * dist) * x012sum[4 * i + 3];
                    sumpx0b += pmn * x012sum[4 * i + 0];
                    sumpx1b += pmn * x012sum[4 * i + 1];
                    sumpx2b += pmn * x012sum[4 * i + 2];
                    sump1b += pmn;
                }
            }
            sumpx0 += sumpx0b;
            sumpx1 += sumpx1b;
            sumpx2 += sumpx2b;
            sump1 += sump1b;
        }

        __syncthreads();
    }

    if (j < m) {
        p1[j] = sump1;
        px[0 * m + j] = sumpx0;
        px[1 * m + j] = sumpx1;
        px[2 * m + j] = sumpx2;
    }
}

__global__ void cpd_sigmaest_cuda(CONST_ARG int m, CONST_ARG int n, float* ctx)
{
    __shared__ float t012[3 * BLOCK_SIZE];

    int thread = threadIdx.x;
    int i = thread + blockIdx.x * BLOCK_SIZE;

    float* x = ctx;
    float* t = x + 3 * n;
    float* temp = t + 3 * m;

    float x0 = 0, x1 = 0, x2 = 0, accum = 0;
    if (i < n) {
        x0 = x[i];
        x1 = x[n + i];
        x2 = x[2 * n + i];
    }

    for (int jb = 0; jb < m; jb += BLOCK_SIZE) {
        int mb = __min(m, jb + BLOCK_SIZE) - jb;

        int jthread = jb + thread;
        t012[3 * thread + 0] = t[0 * m + jthread];
        t012[3 * thread + 1] = t[1 * m + jthread];
        t012[3 * thread + 2] = t[2 * m + jthread];
        
        __syncthreads();

        if (i < n) {
            float accumBlock = 0;
            for (int j = 0; j < mb; j++) {
                float dd0 = x0 - t012[3 * j + 0];
                float dd1 = x1 - t012[3 * j + 1];
                float dd2 = x2 - t012[3 * j + 2];
                float dist = fmaf(dd0, dd0, fmaf(dd1, dd1, dd2 * dd2));
                accumBlock += dist;
            }

            accum += accumBlock;
        }

        __syncthreads();
    }

    if (i < n)
        temp[i] = accum;
}